#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include "conv2d_fp32.h"
#define PLACEHOLDER 1
__global__ void implgemm(param_t param)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z;

    if (x >= param.Oh * param.Ow || y >= param.k || z >= param.n)
        return;

    int oh = x / param.Ow;
    int ow = x % param.Ow;
    int posh_ori = oh * param.u - param.p;
    int posw_ori = ow * param.v - param.q;

    float sum = 0.0;
    int inOffset = z * param.c * param.h * param.w;
    int weiOffset = y * param.c * param.r * param.s;
    int inChannelOffset = param.h * param.w;
    int weightChannelOffset = param.r * param.s;

    for (int i = 0; i < param.r; i++) {
        for (int j = 0; j < param.s; j++) {
            int posh_real = posh_ori + i;
            int posw_real = posw_ori + j;

            if (posh_real >= 0 && posw_real >= 0 && posw_real < param.w && posh_real < param.h) {
                int inOffsetTmp = inOffset + posh_real * param.w + posw_real;
                int weiOffsetTmp = weiOffset;
                for (int channel = 0; channel < param.c; channel++) {
                    sum += param.input[inOffsetTmp] * param.weight[weiOffsetTmp + i * param.s + j];
                    inOffsetTmp += inChannelOffset;
                    weiOffsetTmp += weightChannelOffset;
                }
            }
        }
    }

    int outOffset = z * param.k * param.Oh * param.Ow + y * param.Oh * param.Ow + x;
    param.output[outOffset] = sum;
}
void conv2d_cuda_forward(param_t param)
{
    int threadx = 16;
    int thready = 16;
    int threadz = 1;
    int blockx = ((param.Oh * param.Ow + 15) / 16); 
    int blocky = (param.k + 15) / 16;       
    int blockz = param.n; 
    dim3 block(threadx, thready, threadz);  
    dim3 grid(blockx, blocky, blockz);
    implgemm<<<grid, block>>>(param);
}


__global__ void implgemmbwddata(param_t param)
{
    uint32_t tx = threadIdx.x;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    const uint32_t lane_id = threadIdx.x % 32;
    const uint32_t warp_id = threadIdx.x / 32;
    const uint32_t mma_tid_x = (lane_id / 2) % 8;
    const uint32_t mma_tid_y = (lane_id / 16) * 2 + (lane_id % 2);

    uint32_t weight_lds_addr = (warp_id / 2) * 32 + mma_tid_y * 4;
    uint32_t gradoutput_lds_addr = (warp_id % 2) * 64 + mma_tid_x * 4;

    int x = bx * 128 + gradoutput_lds_addr;
    int y = by * 128 + weight_lds_addr;
    int z = blockIdx.z;

    __shared__ DTYPE smemgradoutput[8 * 128];
    __shared__ DTYPE smemweight[8 * 132];

    DTYPE weight_ldg_reg[4];
    DTYPE gradoutput_ldg_reg[4];

    int posOh_ori[4];
    int posOw_ori[4];
#pragma unroll
    for (int i = 0; i < 4; ++i)
    {
        posOh_ori[i] = ((bx * 128 + tx % 32 + i * 32) / param.w) - (param.r - 1 - param.p);
        posOw_ori[i] = ((bx * 128 + tx % 32 + i * 32) % param.w) - (param.s - 1 - param.q);
    }

    int outOffset = z * param.k * param.Oh * param.Ow;
    int weiC = (by * 128 + tx / 8 * 4);
    int outKOffset = param.Oh * param.Ow;
    int weiCOffset = param.r * param.s;
    int weiKOffset = param.c * param.r * param.s;

    uint32_t weight_sts_addr = (tx % 8) * 132 +
                               (tx / 8) * 4;
    uint32_t gradoutput_sts_addr = (tx / 32) * 128 + (tx % 32);

    DTYPE weight_frag[8];
    DTYPE gradoutput_frag[8];
    DTYPE gradinput_frag[8][8];
#pragma unroll
    for (int i = 0; i < 8; ++i)
    {
#pragma unroll
        for (int j = 0; j < 8; ++j)
        {
            gradinput_frag[i][j] = 0.0f;
        }
    }

    for (int krs = 0; krs < param.r * param.s * param.k; krs += 8)
    {
        int curKRS = krs + tx % 8;
        int rs = param.r * param.s - 1 - curKRS % (param.r * param.s);
        int curK = curKRS / (param.r * param.s);
#pragma unroll
        for (int i = 0; i < 4; ++i)
        {
            if ((curK * param.r * param.s + rs) < param.r * param.s * param.k)
            {
                weight_ldg_reg[i] = param.weight[curK * weiKOffset + (weiC + i) * weiCOffset + rs];
            }
            else
            {
                weight_ldg_reg[i] = 0.0f;
            }
        }
        int curK2 = (krs + tx / 32) / (param.r * param.s);            
        int curR = ((krs + tx / 32) % (param.r * param.s)) / param.s; 
        int curS = ((krs + tx / 32) % (param.r * param.s)) % param.s; 

#pragma unroll
        for (int i = 0; i < 4; ++i)
        {
            int curOh = posOh_ori[i] + curR; 
            int curOw = posOw_ori[i] + curS; 
            int outOffsetTmp = curK2 * outKOffset + curOh * param.Ow + curOw;
            if (curOh >= 0 && curOw >= 0 && curOw < param.Ow && curOh < param.Oh)
            {
                gradoutput_ldg_reg[i] = param.grad_output[outOffset + outOffsetTmp];
            }
            else
            {
                gradoutput_ldg_reg[i] = 0.0f;
            }
        }
        // sts
        for (int i = 0; i < 4; ++i)
        {
            smemweight[weight_sts_addr + i] = weight_ldg_reg[i];
        }
        for (int i = 0; i < 4; ++i)
        {
            smemgradoutput[gradoutput_sts_addr + i * 32] = gradoutput_ldg_reg[i];
        }
        __syncthreads();
#pragma unroll
        for (int subkrs = 0; subkrs < 8; ++subkrs)
        {
#pragma unroll
            for (int i = 0; i < 4; ++i)
            {
                weight_frag[i] = smemweight[weight_lds_addr + subkrs * 132 + i];
                weight_frag[i + 4] = smemweight[weight_lds_addr + subkrs * 132 + i + 16];
            }
#pragma unroll
            for (int i = 0; i < 4; ++i)
            {
                gradoutput_frag[i] = smemgradoutput[gradoutput_lds_addr + subkrs * 128 + i];
                gradoutput_frag[i + 4] = smemgradoutput[gradoutput_lds_addr + subkrs * 128 + i + 32];
            }

#pragma unroll
            for (int i = 0; i < 8; ++i)
            {
#pragma unroll
                for (int j = 0; j < 8; ++j)
                {
                    gradinput_frag[i][j] += (weight_frag[i] * gradoutput_frag[j]);
                }
            }
        }
        __syncthreads();
    }

    int gradinputOffset;
#pragma unroll
    for (int i = 0; i < 4; ++i)
    {
#pragma unroll
        for (int j = 0; j < 4; ++j)
        {
            gradinputOffset = z * param.c * param.h * param.w + (y + i) * param.h * param.w + x + j;
            if (x + j < param.h * param.w && y + i < param.c)
            {
                param.grad_input[gradinputOffset] = gradinput_frag[i][j];
            }
            gradinputOffset = z * param.c * param.h * param.w + (y + i) * param.h * param.w + x + j + 32;
            if (x + j + 32 < param.h * param.w && y + i < param.c)
            {
                param.grad_input[gradinputOffset] = gradinput_frag[i][j + 4];
            }
            gradinputOffset = z * param.c * param.h * param.w + (y + i + 16) * param.h * param.w + x + j;
            if (x + j < param.h * param.w && y + i + 16 < param.c)
            {
                param.grad_input[gradinputOffset] = gradinput_frag[i + 4][j];
            }
            gradinputOffset = z * param.c * param.h * param.w + (y + i + 16) * param.h * param.w + x + j + 32;
            if (x + j + 32 < param.h * param.w && y + i + 16 < param.c)
            {
                param.grad_input[gradinputOffset] = gradinput_frag[i + 4][j + 4];
            }
        }
    }
}
__global__ void implgemmbwdweight(param_t param)
{
    uint32_t tx = threadIdx.x;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    const uint32_t lane_id = threadIdx.x % 32;
    const uint32_t warp_id = threadIdx.x / 32;
    const uint32_t mma_tid_x = (lane_id / 2) % 8;
    const uint32_t mma_tid_y = (lane_id / 16) * 2 + (lane_id % 2);
    uint32_t gradoutput_lds_addr = (warp_id / 2) * 32 + mma_tid_y * 4;
    uint32_t input_lds_addr = (warp_id % 2) * 64 + mma_tid_x * 4;

    int x = bx * 128 + input_lds_addr;
    int y = by * 128 + gradoutput_lds_addr;
    int z = blockIdx.z;

    __shared__ DTYPE smeminput[8 * 128];
    __shared__ DTYPE smemgradoutput[8 * 132];

    int posh_ori[4];
    int posw_ori[4];
#pragma unroll
    for (int i = 0; i < 4; ++i)
    {
        posh_ori[i] = ((bx * 128 + tx % 32 + i * 32) / param.s) - param.p;
        posw_ori[i] = ((bx * 128 + tx % 32 + i * 32) % param.s) - param.q;
    }

    int inOffset = z * param.h * param.w;
    int outK = (by * 128 + tx / 8 * 4);
    int inNOffset = param.c * param.h * param.w;
    int outKOffset = param.Oh * param.Ow;
    int outNOffset = param.k * param.Oh * param.Ow;

    uint32_t gradoutput_sts_addr = (tx % 8) * 132 +
                                   (tx / 8) * 4;
    uint32_t input_sts_addr = (tx / 32) * 128 + (tx % 32);

    DTYPE gradoutput_frag[8];
    DTYPE input_frag[8];
    DTYPE gradweight_frag[8][8];
#pragma unroll
    for (int i = 0; i < 8; ++i)
    {
#pragma unroll
        for (int j = 0; j < 8; ++j)
        {
            gradweight_frag[i][j] = 0.0f;
        }
    }

    for (int nohow = 0; nohow < param.Oh * param.Ow * param.n; nohow += 8)
    {
        int curNOHOW = nohow + tx % 8;
        int ohow = curNOHOW % (param.Oh * param.Ow);
        int curN_1 = curNOHOW / (param.Oh * param.Ow);
#pragma unroll
        for (int i = 0; i < 4; ++i)
        {
            if (curNOHOW < param.Oh * param.Ow * param.n)
            {
                smemgradoutput[gradoutput_sts_addr + i] = param.grad_output[curN_1 * outNOffset + (outK + i) * outKOffset + ohow];
            }
            else
            {
                smemgradoutput[gradoutput_sts_addr + i] = 0.0f;
            }
        }

        int curN_2 = (nohow + tx / 32) / (param.Oh * param.Ow);             
        int curOh = ((nohow + tx / 32) % (param.Oh * param.Ow)) / param.Ow; 
        int curOw = ((nohow + tx / 32) % (param.Oh * param.Ow)) % param.Ow; 

#pragma unroll
        for (int i = 0; i < 4; ++i)
        {
            int curH = posh_ori[i] + curOh; 
            int curW = posw_ori[i] + curOw;
            int inOffsetTmp = curN_2 * inNOffset + curH * param.w + curW;
            if (curH >= 0 && curW >= 0 && curW < param.w && curH < param.h)
            {
                smeminput[input_sts_addr + i * 32] = param.input[inOffset + inOffsetTmp];
            }
            else
            {
                smeminput[input_sts_addr + i * 32] = 0.0f;
            }
        }
        __syncthreads();
#pragma unroll
        for (int subnohow = 0; subnohow < 8; ++subnohow)
        {
#pragma unroll
            for (int i = 0; i < 4; ++i)
            {
                gradoutput_frag[i] = smemgradoutput[gradoutput_lds_addr + subnohow * 132 + i];
                gradoutput_frag[i + 4] = smemgradoutput[gradoutput_lds_addr + subnohow * 132 + i + 16];
            }
#pragma unroll
            for (int i = 0; i < 4; ++i)
            {
                input_frag[i] = smeminput[input_lds_addr + subnohow * 128 + i];
                input_frag[i + 4] = smeminput[input_lds_addr + subnohow * 128 + i + 32];
            }

#pragma unroll
            for (int i = 0; i < 8; ++i)
            {
#pragma unroll
                for (int j = 0; j < 8; ++j)
                {
                    gradweight_frag[i][j] += (gradoutput_frag[i] * input_frag[j]);
                }
            }
        }
        __syncthreads();
    }

    int gradweightoffset;
#pragma unroll
    for (int i = 0; i < 4; ++i)
    {
#pragma unroll
        for (int j = 0; j < 4; ++j)
        {
            gradweightoffset = z * param.r * param.s + (y + i) * param.c * param.r * param.s + x + j;
            if (x + j < param.r * param.s && y + i < param.k)
            {
                param.grad_weight[gradweightoffset] = gradweight_frag[i][j];
            }
            gradweightoffset = z * param.r * param.s + (y + i) * param.c * param.r * param.s + x + j + 32;
            if (x + j + 32 < param.r * param.s && y + i < param.k)
            {
                param.grad_weight[gradweightoffset] = gradweight_frag[i][j + 4];
            }
            gradweightoffset = z * param.r * param.s + (y + i + 16) * param.c * param.r * param.s + x + j;
            if (x + j < param.r * param.s && y + i + 16 < param.k)
            {
                param.grad_weight[gradweightoffset] = gradweight_frag[i + 4][j];
            }
            gradweightoffset = z * param.r * param.s + (y + i + 16) * param.c * param.r * param.s + x + j + 32;
            if (x + j + 32 < param.r * param.s && y + i + 16 < param.k)
            {
                param.grad_weight[gradweightoffset] = gradweight_frag[i + 4][j + 4];
            }
        }
    }
}

void conv2d_cuda_backward(param_t param)
{
    int blockx = ((param.h * param.w + 127) / 128); 
    int blocky = (param.c + 127) / 128;       
    int blockz = param.n;                     // blockz  number
    // 合并threadx与thready
    int threadx = 256; // threadx number per block
    int thready = 1;   // thready number per block
    int threadz = 1;   // threadz number per block
    dim3 blockbwddata(threadx, thready, threadz);
    dim3 gridbwddata(blockx, blocky, blockz);
    implgemmbwddata<<<gridbwddata, blockbwddata>>>(param);

    blockx = (param.r * param.s + 127) / 128; // blockx  number
    blocky = (param.k + 127) / 128;     // blocky  number
    blockz = param.c;                   // blockz  number
    // 合并threadx与thready
    threadx = 256; // threadx number per block
    thready = 1;   // thready number per block
    threadz = 1;   // threadz number per block
    dim3 blockbwdweight(threadx, thready, threadz);
    dim3 gridbwdweight(blockx, blocky, blockz);
    implgemmbwdweight<<<gridbwdweight, blockbwdweight>>>(param);
}